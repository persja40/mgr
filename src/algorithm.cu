#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>

#include "read_iris.h"

__global__ void kernel_gpu(float *data, int dim, float *answer)
{
    if (threadIdx.x < dim)
    {
        atomicAdd(answer, data[threadIdx.x] * 2);
    }
}

int main(int argc, char **argv)
{
    // auto t = read_iris();
    // std::cout << "Algorithm " << t.size() << std::endl;
    std::vector<float> test{0, 1, 2, 3, 4};

    float *d_answer;
    hipMallocManaged(&d_answer, sizeof(float));
    *d_answer = 0.0;
    float *d_test;
    hipMalloc(&d_test, test.size() * sizeof(float));

    float answer = 123.456;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "SET: " << answer << std::endl;

    hipMemcpy(d_test, test.data(), test.size() * sizeof(float), hipMemcpyHostToDevice);

    kernel_gpu<<<1, 50>>>(d_test, test.size(), d_answer);
    hipDeviceSynchronize();

    answer = 989.123;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "GPU: " << answer << std::endl;
    std::cout << "CPU: " << std::accumulate(begin(test), end(test), 0) << std::endl;
    hipFree(d_test);
}