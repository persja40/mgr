#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <math.h>
#include <stdio.h>

#include "read_iris.h"

/*
------------------------------------------------------------------------------------
    GPU
------------------------------------------------------------------------------------
*/

__global__ void xTx(float *data, int dim, float *answer)
{
    if (threadIdx.x < dim)
    {
        atomicAdd(answer, data[threadIdx.x] * 2);
        printf("Hello cuda thread %d\n", threadIdx.x);
    }
}

__global__ void kernel_gpu(float *data, int dim, float *answer)
{
    float *mult;
    hipMalloc(&mult, sizeof(float));
    *mult = 0.0;
    printf("inside cuda %f\n", *mult);
    xTx<<<1, dim>>>(data, dim, mult);
    hipDeviceSynchronize();
    printf("inside cuda %f\n", *mult);
    // expf single precision
    *answer = expf(-0.5 * (*mult)) / (2 * pow(M_PI, dim * 0.5));
    hipFree(mult);
    printf("kernel\n");
}

/*
------------------------------------------------------------------------------------
    CPU
------------------------------------------------------------------------------------
*/

int main(int argc, char **argv)
{
    // auto t = read_iris();
    // std::cout << "Algorithm " << t.size() << std::endl;
    std::vector<float> test{0, 1, 2, 3, 4};

    float *d_answer;
    hipMalloc(&d_answer, sizeof(float));
    float *d_test;
    hipMalloc(&d_test, test.size() * sizeof(float));

    float answer = 0.0;
    hipMemcpy(d_answer, &answer, sizeof(answer), hipMemcpyHostToDevice);

    // answer = 123.456;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "SET: " << answer << std::endl;

    hipMemcpy(d_test, test.data(), test.size() * sizeof(float), hipMemcpyHostToDevice);

    // xTx<<<1, test.size()>>>(d_test, test.size(), d_answer);
    hipDeviceSynchronize();

    kernel_gpu<<<1, 1>>>(d_test, test.size(), d_answer);
    hipDeviceSynchronize();

    answer = 989.123;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "GPU: " << answer << std::endl;

    auto cpu_sum = [](float a, float b) {
        return a + 2 * b;
    };
    std::cout << "CPU: " << std::accumulate(begin(test), end(test), 0, cpu_sum) << std::endl;
    hipFree(d_test);
    hipFree(d_answer);
}