#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <math.h>

#include "read_iris.h"

__global__ void xTx(float *data, int dim, float *answer)
{
    if (threadIdx.x < dim)
    {
        atomicAdd(answer, data[threadIdx.x] * 2);
    }
}

__global__ void kernel_gpu(float *data, int dim, float *answer)
{
    float *mult = new float;
    *mult = 0.0;
    xTx<<<1, dim>>>(data, dim, mult);
    printf("inside cuda %f",*mult);
    // expf single precision
    *answer = expf(-0.5 * (*mult)) / (2 * pow(M_PI, dim * 0.5));
}

int main(int argc, char **argv)
{
    // auto t = read_iris();
    // std::cout << "Algorithm " << t.size() << std::endl;
    std::vector<float> test{0, 1, 2, 3, 4};

    float *d_answer;
    hipMallocManaged(&d_answer, sizeof(float));
    *d_answer = 0.0;
    float *d_test;
    hipMalloc(&d_test, test.size() * sizeof(float));

    float answer = 123.456;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "SET: " << answer << std::endl;

    hipMemcpy(d_test, test.data(), test.size() * sizeof(float), hipMemcpyHostToDevice);

    kernel_gpu<<<1, 1>>>(d_test, test.size(), d_answer);
    hipDeviceSynchronize();

    answer = 989.123;
    hipMemcpy(&answer, d_answer, sizeof(answer), hipMemcpyDeviceToHost);
    std::cout << "GPU: " << answer << std::endl;
    std::cout << "CPU: " << std::accumulate(begin(test), end(test), 0) << std::endl;
    hipFree(d_test);
}