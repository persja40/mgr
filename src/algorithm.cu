#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <math.h>
#include <stdio.h>
#include <cmath>
#include <tuple>


#include "read_iris.h"

/*
------------------------------------------------------------------------------------
    GPU
------------------------------------------------------------------------------------
*/

__global__ void g_h_sum(float *data, int m, int n, float h, float *answer)
{
    if (blockIdx.x < m * m)
        if (threadIdx.x < n)
        {
            // std::printf("Hello %d %d \n", blockIdx.x, threadIdx.x);
            int j = blockIdx.x % m;
            int i = blockIdx.x / m;
            __shared__ float *tmp_vec;
            if (threadIdx.x == 0)
                hipMalloc(&tmp_vec, n * sizeof(float));
            __syncthreads();

            tmp_vec[threadIdx.x] = (data[threadIdx.x + j * n] - data[threadIdx.x + i * n]) / h;
            __syncthreads();

            __shared__ float xTx;
            if (threadIdx.x == 0)
                xTx = 0.0;
            __syncthreads();

            // std::printf("Hello %d %d %f\n", blockIdx.x, threadIdx.x, xTx);
            // __syncthreads();

            atomicAdd(&xTx, tmp_vec[threadIdx.x] * tmp_vec[threadIdx.x]);
            __syncthreads();

            if (threadIdx.x == 0)
            {
                xTx = expf(-0.25 * xTx) / pow(4 * M_PI, n * 0.5) - 2 * expf(-0.5 * xTx) / (2 * pow(M_PI, n * 0.5));
                atomicAdd(answer, xTx);
            }

            __syncthreads();
            if (threadIdx.x == 0){
                hipFree(tmp_vec);
                // std::printf("Hello %d %d answer:%f\n", blockIdx.x, threadIdx.x, xTx);
                // std::printf("Hello %d %d answer:%f\n", blockIdx.x, threadIdx.x, *answer);
            }
        }
}

int main(int argc, char **argv)
{
    auto tpl = read_iris_gpu();    
    int m = std::get<1>(tpl);
    int n = std::get<2>(tpl);

    auto& t = std::get<0>(tpl);

    const float *ptr = t.data();
    // for(int i=0; i<m; i++)
    //     std::cout<<ptr[i*n]<<" "<<ptr[i*n+1]<<" "<<ptr[i*n+2]<<" "<<ptr[i*n+3]<<std::endl;

    float *d_answer;
    hipMalloc(&d_answer, sizeof(float));
    float answer = 0.0;
    hipMemcpy(d_answer, &answer, sizeof(float), hipMemcpyHostToDevice);

    float *d_t;
    hipMalloc(&d_t, m * n * sizeof(float));
    hipMemcpy(d_t, t.data(), m * n * sizeof(float), hipMemcpyHostToDevice);

    g_h_sum<<<m * m, n>>>(d_t, m, n, 1.0, d_answer);
    hipDeviceSynchronize();

    answer = 989.123;
    hipMemcpy(&answer, d_answer, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "GPU: " << answer << std::endl;

    hipFree(d_t);
    hipFree(d_answer);
}