#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <math.h>
#include <stdio.h>
#include <cmath>
#include <thread>
#include <future>
#include <atomic>

#include "read_iris.h"

/*
------------------------------------------------------------------------------------
    GPU
------------------------------------------------------------------------------------
*/

__global__ void g_h_sum(float *data, int m, int n, float h, float *answer)
{
    if (blockIdx.x < m * m)
        if (threadIdx.x < n)
        {
            std::printf("Hello %d %d \n", blockIdx.x, threadIdx.x);
            int j = blockIdx.x % m;
            int i = blockIdx.x / m;
            __shared__ float *tmp_vec;
            if (threadIdx.x == 0)
                hipMalloc(&tmp_vec, n * sizeof(float));
            __syncthreads();

            tmp_vec[threadIdx.x] = (data[threadIdx.x + j * n] - data[threadIdx.x + i * n]) / h;
            __syncthreads();

            __shared__ float xTx;
            if (threadIdx.x == 0)
                xTx = 1.0;
            __syncthreads();

            // std::printf("Hello %d %d %f\n", blockIdx.x, threadIdx.x, xTx);
            // __syncthreads();

            atomicAdd(&xTx, tmp_vec[threadIdx.x] * tmp_vec[threadIdx.x]);
            __syncthreads();

            if (threadIdx.x == 0)
            {
                xTx = expf(-0.25 * xTx) / pow(4 * M_PI, n * 0.5) - 2 * expf(-0.5 * xTx) / (2 * pow(M_PI, n * 0.5));
                atomicAdd(answer, xTx);
            }

            __syncthreads();
            if (threadIdx.x == 0){
                hipFree(tmp_vec);
                std::printf("Hello %d %d answer:%f\n", blockIdx.x, threadIdx.x, xTx);
                std::printf("Hello %d %d answer:%f\n", blockIdx.x, threadIdx.x, *answer);
            }
        }
}

int main(int argc, char **argv)
{
    auto t = read_iris();

    for(const auto e:t[0])
        std::cout<<e<<"\t";
    std::cout<<std::endl;

    std::cout << "t size: " << t.size() << std::endl;
    // std::vector<float> test{0, 1, 2, 3, 4};
    int m = t.size();
    int n = t[0].size();

    float *d_answer;
    hipMalloc(&d_answer, sizeof(float));
    float answer = 0.0;
    hipMemcpy(d_answer, &answer, sizeof(float), hipMemcpyHostToDevice);

    float *d_t;
    hipMalloc(&d_t, m * n * sizeof(float));
    hipMemcpy(d_t, t.data(), m * n * sizeof(float), hipMemcpyHostToDevice);

    g_h_sum<<<m * m, n>>>(d_t, m, n, 2.0, d_answer);
    hipDeviceSynchronize();

    answer = 989.123;
    hipMemcpy(&answer, d_answer, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "GPU: " << answer << std::endl;

    hipFree(d_t);
    hipFree(d_answer);
}